#include<stdio.h>
#include "hip/hip_runtime.h"

#include <cstdlib>

const long int INF = 99999999;
const int N = 4;

__global__ void ComputeMinPath(int *d_Matrix) {
	int row = blockIdx.x;
	int col = threadIdx.x;

	for (int j = 0; j < N; j++) {

		d_Matrix[row * N + col] = d_Matrix[row * N + j] + d_Matrix[j * N + col] < d_Matrix[row * N + col] ? 
			d_Matrix[row * N + j] + d_Matrix[j * N + col]
		: d_Matrix[row * N + col];
	}
}

void RoyFloyd(int *h_Matrix, int N) {

	size_t size = N * N * sizeof(int);
	int *d_Matrix;

	hipMalloc(&d_Matrix, size);
	hipMemcpy(d_Matrix, h_Matrix, size, hipMemcpyHostToDevice);

	for (int k = 0; k < N; k++) {
		ComputeMinPath<<<N, N>>>(d_Matrix);
	}

	hipMemcpy(h_Matrix, d_Matrix, size, hipMemcpyDeviceToHost);

	hipFree(d_Matrix);
}

int main() {

	int matrix[N*N] = { 
		0,   5,  INF, 10, 
		INF,  0,  3,  INF,
		INF, INF, 0,   1 ,
		INF, INF, INF, 0
	};

	RoyFloyd(matrix, N);

	for (int i = 0; i < N * N; i++) {
		if(matrix[i] == INF)
		{
			printf("INF  ");
		}
		else {
			printf("%d  ", matrix[i]);
		}
		if ((i + 1) % N == 0) {
			printf("\n");
		}
	}

	getchar();
	return 0;
}